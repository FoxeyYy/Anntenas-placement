/**
 * Parallel computing (2015-2016 course)
 *
 * Antennas setup
 * CUDA implementation
 *
 * @author Hector Del Campo Pando
 * @author Alberto Gutierrez Perez
 */

// Includes generales
#include <stdlib.h>
#include <stdio.h>
#include <limits.h>


// Include para las utilidades de computación paralela
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

/**
 * Estructura antena
 */
typedef struct {
	int y;
	int x;
} Antena;

/**
 * Estructura para guardar la informacion del numero maximo global
 */
typedef struct {
	int max;
	int pos;
} Max_data;


/**
 * Macros para acceder al maximo y su posicion
 */
#define valor(m) m->max
#define pos(max) max->pos

/**
 * Macro para acceder a las posiciones del mapa
 */
#define m(y,x) mapa[ (y * cols) + x ]
#define posicion(y,x) (y * cols) + x
#define row(n)	((int)n) / cols
#define col(n)	n % cols

/**
 * Macro para la funcion manhattan
 */
#define manhattan(a, i, j) (abs(a.x -j) + abs(a.y - i)) * (abs(a.x -j) + abs(a.y - i))
#define nueva_antena(n)	{row(n), col(n)}

#define posicion_thread() (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.x + threadIdx.y * blockDim.x)

/**
 * Macros para la reduccion
 */
#define MAX_THREADS 1024
#define NUM_THREADS_BLOCK 256
#define NUM_BLOCKS ((int)rows*cols/NUM_THREADS_BLOCK + 1)

__global__ void iniciarMapa(int *mapa, int rows, int cols){
	int posicion = posicion_thread();
	if(posicion < rows*cols)
		mapa[posicion] = INT_MAX;
}


template <unsigned int blockSize>
__global__ void max_kernel(int *entrada_max, int *entrada_pos, int *salida_max, int *salida_pos, int size){
	extern __shared__ int s[];

	unsigned int nHilos = blockDim.x;
	unsigned int id = threadIdx.x;	//id thread en bloque
	unsigned int idBloque = blockIdx.x;	//id del bloque en grid
	unsigned int posicion = id + idBloque*nHilos;	//Posicion en el mapa

	int *maximos = (int*)s;
	int *posiciones = (int*)&maximos[nHilos];

	int myMax = INT_MIN, pos = INT_MAX;

	if(entrada_pos == NULL){
		while(posicion < size){
			if(entrada_max[posicion] > myMax){
				myMax = entrada_max[posicion];
				pos = posicion;
			}
			posicion+=(nHilos*gridDim.x);
		}
	}else{
		while(posicion < size){
			if(entrada_max[posicion] > myMax){
				myMax = entrada_max[posicion];
				pos = entrada_pos[posicion];
			}
			posicion+=(nHilos*gridDim.x);
		}
	}

	maximos[id] = myMax;
	posiciones[id] = pos;
	__syncthreads();



	if(blockSize>= 1024){
		if( id < 512)
			if(maximos[id + 512] > maximos[id] || (maximos[id + 512] == maximos[id] && posiciones[id+512] < posiciones[id])){
				maximos[id] = maximos[id+512];
				posiciones[id] = posiciones[id+512];
			}
			__syncthreads();
	}
	if(blockSize>= 512){
		if( id < 256)
			if(maximos[id + 256] > maximos[id] || (maximos[id + 256] == maximos[id] && posiciones[id+256] < posiciones[id])){
				maximos[id] = maximos[id+256];
				posiciones[id] = posiciones[id+256];
			}
			__syncthreads();
	}
	if(blockSize>= 256){
		if( id < 128)
			if(maximos[id + 128] > maximos[id] || (maximos[id + 128] == maximos[id] && posiciones[id+128] < posiciones[id])){
				maximos[id] = maximos[id+128];
				posiciones[id] = posiciones[id+128];
			}			
		__syncthreads();
	}
	if(blockSize>= 128){
		if( id < 64)
			if(maximos[id + 64] > maximos[id] || (maximos[id + 64] == maximos[id] && posiciones[id+64] < posiciones[id])){
				maximos[id] = maximos[id+64];
				posiciones[id] = posiciones[id+64];
			}
			__syncthreads();
	}

	if(id < 32){
		if(blockSize>= 64){	if((maximos[id + 32] > maximos[id] || (maximos[id + 32] == maximos[id] && posiciones[id+32] < posiciones[id]))){maximos[id] = maximos[id+32]; posiciones[id] = posiciones[id+32];}}
		if(blockSize>=32){	if((maximos[id + 16] > maximos[id] || (maximos[id + 16] == maximos[id] && posiciones[id+16] < posiciones[id]))){maximos[id] = maximos[id+16]; posiciones[id] = posiciones[id+16];}}
		if(blockSize>=16){	if((maximos[id + 8] > maximos[id] || (maximos[id + 8] == maximos[id] && posiciones[id+8] < posiciones[id]))){maximos[id] = maximos[id+8]; posiciones[id] = posiciones[id+8];}}
		if(blockSize>=8){	if((maximos[id + 4] > maximos[id] || (maximos[id + 4] == maximos[id] && posiciones[id+4] < posiciones[id]))){maximos[id] = maximos[id+4]; posiciones[id] = posiciones[id+4];}}
		if(blockSize>=4){	if((maximos[id + 2] > maximos[id] || (maximos[id + 2] == maximos[id] && posiciones[id+2] < posiciones[id]))){maximos[id] = maximos[id+2]; posiciones[id] = posiciones[id+2];}}
		if(blockSize>=2){	if((maximos[id + 1] > maximos[id] || (maximos[id + 1] == maximos[id] && posiciones[id+1] < posiciones[id]))){maximos[id] = maximos[id+1]; posiciones[id] = posiciones[id+1];}}
	}

	if(id == 0){
		salida_max[idBloque] = maximos[0];
		salida_pos[idBloque] = posiciones[0];
	}

}

__global__ void actualizar_kernel( int *mapa, Antena antena, int rows, int cols){
	m(antena.y, antena.x) = 0;
	int nuevadist;
	unsigned int desplHor = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int desplVer = threadIdx.y + blockIdx.y*blockDim.y;
	int j;

	for(int i = antena.y - desplVer; i >= 0; i-=blockDim.y){
		j = antena.x + desplHor;
		nuevadist = manhattan(antena,i,j);
		if(nuevadist > m(i,j))	break;
		for(; j < cols; j+=blockDim.x){
			nuevadist = manhattan(antena,i,j);
			if(nuevadist > m(i,j))	break;
			m(i,j) = nuevadist;
		}
	}

	for(int i = antena.y + desplVer; i < rows; i+=blockDim.y){
		j = antena.x - desplHor;
		nuevadist = manhattan(antena,i,j);
		if(nuevadist > m(i,j))	break;
		for(; j >= 0; j-=blockDim.x){
			nuevadist = manhattan(antena,i,j);
			if(nuevadist > m(i,j))	break;
			m(i,j) = nuevadist;
		}
	}

	for(int i = antena.y + desplVer; i < rows; i+=blockDim.y){
		j = antena.x + desplHor;
		nuevadist = manhattan(antena,i,j);
		if(nuevadist > m(i,j))	break;
		for(; j < cols; j+=blockDim.x){
			nuevadist = manhattan(antena,i,j);
			if(nuevadist > m(i,j))	break;
			m(i,j) = nuevadist;
		}
	}

	for(int i = antena.y - desplVer; i >= 0; i-=blockDim.y){
		j = antena.x - desplHor;
		nuevadist = manhattan(antena,i,j);
		if(nuevadist > m(i,j))	break;
		for(; j >= 0; j-=blockDim.x){
			nuevadist = manhattan(antena,i,j);
			if(nuevadist > m(i,j))	break;
			m(i,j) = nuevadist;
		}
	}
}

/**
 * Función de ayuda para imprimir el mapa
 */
void print_mapa(int * mapa, int rows, int cols, Antena * a){


	if(rows > 50 || cols > 30){
		printf("Mapa muy grande para imprimir\n");
		return;
	};

	#define ANSI_COLOR_RED     "\x1b[31m"
	#define ANSI_COLOR_GREEN   "\x1b[32m"
	#define ANSI_COLOR_RESET   "\x1b[0m"

	printf("Mapa [%d,%d]\n",rows,cols);
	for(int i=0; i<rows; i++){
		for(int j=0; j<cols; j++){

			int val = m(i,j);

			if(val == 0){
				if(a != NULL && a->x == j && a->y == i){
					printf( ANSI_COLOR_RED "   A"  ANSI_COLOR_RESET);
				} else { 
					printf( ANSI_COLOR_GREEN "   A"  ANSI_COLOR_RESET);
				}
			} else {
				printf("%4d",val);
			}
		}
		printf("\n");
	}
	printf("\n");
}

/**
 * Funcion depuracion para CUDA
 */
void print_mapa_cuda(int *mapa, int rows, int cols){
		int *m = (int*)malloc(sizeof(int)*rows*cols);
		hipMemcpy(m, mapa, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
		print_mapa(m,rows,cols,NULL);
		free(m);
		getchar();
}

/**
 * Actualizar el mapa con la nueva antena
 */
void actualizar(int *mapa, Antena antena, int rows, int cols){
	dim3 hilos(6,6);
	dim3 bloques(32);
	actualizar_kernel<<<bloques, hilos>>>(mapa,antena,rows,cols);
}



/**
 * Calcular la distancia máxima en el mapa
 */
Max_data calcular_max(int * mapa, int *maximos, int *posiciones, int rows, int cols){
	int posicion;
	int maximo;
	int aux = NUM_BLOCKS, nBloques = NUM_BLOCKS;

	//reduccion por bloques
	max_kernel<NUM_THREADS_BLOCK><<<nBloques, NUM_THREADS_BLOCK, sizeof(int)*NUM_THREADS_BLOCK*2>>>(mapa, NULL, maximos, posiciones,rows*cols);

	//reduccion de los bloques
	nBloques = (nBloques - 1)/NUM_THREADS_BLOCK + 1;

	while(aux != 1){
		max_kernel<NUM_THREADS_BLOCK><<<nBloques, NUM_THREADS_BLOCK, sizeof(int)*2*NUM_THREADS_BLOCK>>>(maximos, posiciones, maximos, posiciones, aux);
		aux = nBloques;
		nBloques = (nBloques - 1)/NUM_THREADS_BLOCK + 1;
	}
	//Fin reduccion bloques

	hipMemcpy(&maximo, &maximos[0], sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&posicion, &posiciones[0], sizeof(int), hipMemcpyDeviceToHost);

	Max_data max_global = {maximo, posicion};
	return max_global;
}


/**
 * Función principal
 */
int main(int nargs, char ** vargs){

	//
	// 1. LEER DATOS DE ENTRADA
	//

	// Comprobar número de argumentos
	if(nargs < 7){
		fprintf(stderr,"Uso: %s rows cols distMax nAntenas x0 y0 [x1 y1, ...]\n",vargs[0]);
		return -1;
	}

	// Leer los argumentos de entrada
	int rows = atoi(vargs[1]);
	int cols = atoi(vargs[2]);
	int distMax = atoi(vargs[3]);
	int nAntenas = atoi(vargs[4]);

	if(nAntenas<1 || nargs != (nAntenas*2+5)){
		fprintf(stderr,"Error en la lista de antenas\n");
		return -1;
	}


	// Mensaje
	printf("Calculando el número de antenas necesarias para cubrir un mapa de"
	   " (%d x %d)\ncon una distancia máxima no superior a %d "
	   "y con %d antenas iniciales\n\n",rows,cols,distMax,nAntenas);

	// Reservar memoria para las antenas
	Antena *antenas = (Antena*)malloc(sizeof(Antena) * (size_t) nAntenas);
	if(!antenas){
		fprintf(stderr,"Error al reservar memoria para las antenas inicales\n");
		return -1;
	}	
	
	// Leer antenas
	for(int i=0; i<nAntenas; i++){
		antenas[i].x = atoi(vargs[5+i*2]);
		antenas[i].y = atoi(vargs[6+i*2]);

		if(antenas[i].y<0 || antenas[i].y>=rows || antenas[i].x<0 || antenas[i].x>=cols ){
			fprintf(stderr,"Antena #%d está fuera del mapa\n",i);
			return -1;
		}
	}


	//
	// 2. INICIACIÓN
	//

	// Medir el tiempo
	clock_t reloj = clock();
	double tiempo;
	hipSetDevice(0);

	// Crear el mapa
	int * mapa;
	hipMalloc((void**) &mapa , (rows*cols) * sizeof(int) );

	// Iniciar el mapa con el valor MAX INT
	iniciarMapa<<<NUM_BLOCKS, NUM_THREADS_BLOCK>>>(mapa,rows,cols);

	// Colocar las antenas iniciales
	for(int i=0; i<nAntenas; i++){
		actualizar(mapa, antenas[i], rows, cols);
	}

	// Debug
#ifdef DEBUG
	print_mapa(mapa,rows,cols,NULL);
#endif


	//
	// 3. CALCULO DE LAS NUEVAS ANTENAS
	//

	// Contador de antenas
	int nuevas = 0;
	Max_data max;

	// Variables para CUDA
	int nBloques = NUM_BLOCKS;
	
	int *posiciones;
	int *maximos;
	hipMalloc((void**) &posiciones, nBloques * sizeof(int));
	hipMalloc((void**) &maximos, nBloques * sizeof(int));

	while(1){

		// Calcular el máximo
		max = calcular_max(mapa,maximos,posiciones,rows,cols);

		// Salimos si ya hemos cumplido el maximo
		if (max.max <= distMax) break;	

		// Incrementamos el contador
		nuevas++;
		
		// Calculo de la nueva antena y actualización del mapa
		Antena antena = nueva_antena(max.pos);
		actualizar(mapa,antena,rows,cols);

	}

	reloj = clock() - reloj;

	// Debug
#ifdef DEBUG
	print_mapa(mapa,rows,cols,NULL);
#endif
	
	//Liberar recursos en el host
	hipFree(mapa);
	hipFree(posiciones);
	hipFree(maximos);

	hipDeviceReset();

	//
	// 4. MOSTRAR RESULTADOS
	//

	// tiempo
	tiempo = (double)reloj / CLOCKS_PER_SEC ;

	// Salida
	printf("Result: %d\n",nuevas);
	printf("Time: %f\n",tiempo);


	return 0;
}




